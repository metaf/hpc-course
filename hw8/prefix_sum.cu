#include "hip/hip_runtime.h"
/**
 * University of Pittsburgh
 * Department of Computer Science
 * CS1645: Introduction to HPC Systems
 * Instructor Bryan Mills, PhD
 * This is a skeleton for implementing prefix sum using GPU, inspired
 * by nvidia course of similar name.
 */

#include <stdio.h>
#include "timer.h"
#include <math.h>
#include <string.h>

#define N 512

/*
 * You should implement the simple scan function here!
 */
__global__ void scan_simple(float *g_odata, float *g_idata, int n) {
	int tid=threadIdx.x;
	extern  __shared__  float sharedmem[]; //this is 2N floats
	float* s_in = (float *) sharedmem;
	float* s_out = (float*) &s_in[n];
	float* temp; //for swapping

	s_out[tid]=g_idata[tid] //TODO: do we need to shift here?
	//NOTE: here, we read into OUT not IN, because in our loop we have to swap them.

	__syncthreads()
	for (int offset = 1; offset < n; offset *=2){
		temp = s_in;
		s_in = s_out;
		s_out = temp;
		if(tid >= offset){
			s_out[tid] = s_in[tid - offset] + s_in[tid];
		}
		else{
			s_out[tid] = s_in[tid];
		}
		__syncthreads()
	}
	//NOTE: we swap buffer pointers at the beginning of the loop. so at the end of the
	// 			last iteration we'll have the final values in s_out

	// STUDENT: YOUR CODE GOES HERE.
	g_odata[tid] = s_out[tid];

}

/*
 * You should implement the prescan kernel function here!
 */
__global__ void prescan(float *g_odata, float *g_idata, int n) {
	extern  __shared__  float temp[];

	// STUDENT: YOUR CODE GOES HERE.

}

/*
 * Fills an array a with n random floats.
 */
void random_floats(float* a, int n) {
	float d;
	// Comment out this line if you want consistent "random".
	srand(time(NULL));
	for (int i = 0; i < n; ++i) {
		d = rand() % 8;
		a[i] = ((rand() % 64) / (d > 0 ? d : 1));
	}
}

/*
 * Simple Serial implementation of scan.
 */
void serial_scan(float* out, float* in, int n) {
	float total_sum = 0;
	out[0] = 0;
	for (int i = 1; i < n; i++) {
		total_sum += in[i-1];
		out[i] = out[i-1] + in[i-1];
	}
	if (total_sum != out[n-1]) {
		printf("Warning: exceeding accuracy of float.\n");
	}
}

/*
 * This is a simple function that confirms that the output of the scan
 * function matches that of a golden image (array).
 */
bool printError(float *gold_out, float *test_out, bool show_all) {
	bool firstFail = true;
	bool error = false;
	float epislon = 0.1;
	float diff = 0.0;
	for (int i = 0; i < N; ++i) {
		diff = abs(gold_out[i] - test_out[i]);
		if ((diff > epislon) && firstFail) {
			printf("ERROR: gold_out[%d] = %f != test_out[%d] = %f // diff = %f \n", i, gold_out[i], i, test_out[i], diff);
			firstFail = show_all;
			error = true;
		}
	}
	return error;
}

int main(void) {
	float *in, *out, *gold_out; // host
	float *d_in, *d_out; // device
	int size = sizeof(float) * N;

	timerStart();
	hipMalloc((void **)&d_in, size);
	hipMalloc((void **)&d_out, size);

	in = (float *)malloc(size);
	random_floats(in, N);
	out = (float *)malloc(size);
	gold_out = (float *)malloc(size);
	printf("TIME: Init took %d ms\n",  timerStop());
	// ***********
	// RUN SERIAL SCAN
	// ***********
	timerStart();
	serial_scan(gold_out, in, N);
	printf("TIME: Serial took %d ms\n",  timerStop());

	timerStart();
	hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
	printf("TIME: Copy took %d ms\n",  timerStop());
	// ***********
	// RUN SIMPLE SCAN
	// ***********
	timerStart();
	scan_simple<<< 1, 512, N * 2 * sizeof(float)>>>(d_out, d_in, N);
	//1 block, 512 threads, 2N *Float arrays in shared memory
	hipDeviceSynchronize();
	printf("TIME: Simple kernel took %d ms\n",  timerStop());
	timerStart();
	hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
	printf("TIME: Copy back %d ms\n",  timerStop());

	if (printError(gold_out, out, false)) {
		printf("ERROR: The simple scan function failed to produce proper output.\n");
	} else {
		printf("CONGRATS: The simple scan function produced proper output.\n");
	}

	// ***********
	// RUN PRESCAN
	// note size change in number of threads, only need 256 because each
	// thread should handle 2 elements.
	// ***********
	timerStart();
	prescan<<< 1, 256, N * 2 * sizeof(float)>>>(d_out, d_in, N);
	hipDeviceSynchronize();
	printf("TIME: Prescan kernel took %d ms\n",  timerStop());
	timerStart();
	hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
	printf("TIME: Copy back %d ms\n",  timerStop());

	if (printError(gold_out, out, false)) {
		printf("ERROR: The prescan function failed to produce proper output.\n");
	} else {
		printf("CONGRATS: The prescan function produced proper output.\n");
	}

	return 0;
}
